#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cmath>

#include "kernel.h"

#include <gmp.h>
#include "cgbn/cgbn.h"

#define THREADS_PER_BLOCK 128

#define CGBN_CHECK(report) cgbn_check(report, __FILE__, __LINE__)

template<uint32_t tpi, uint32_t bits>
class pollard_params_t {
public:
    static const uint32_t TPI = tpi;                   // threads per instance
    static const uint32_t BITS = bits;                 // instance size
};

template<class params>
struct factor_result_t {
    cgbn_mem_t<params::BITS> factor;
    unsigned b;
};

void cgbn_check(cgbn_error_report_t *report, const char *file = nullptr, int32_t line = 0) {
    // check for cgbn errors

    if (cgbn_error_report_check(report)) {
        printf("\n");
        printf("CGBN error occurred: %s\n", cgbn_error_string(report));

        if (report->_instance != 0xFFFFFFFF) {
            printf("Error reported by instance %d", report->_instance);
            if (report->_blockIdx.x != 0xFFFFFFFF || report->_threadIdx.x != 0xFFFFFFFF)
                printf(", ");
            if (report->_blockIdx.x != 0xFFFFFFFF)
                printf("blockIdx=(%d, %d, %d) ", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
            if (report->_threadIdx.x != 0xFFFFFFFF)
                printf("threadIdx=(%d, %d, %d)", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
            printf("\n");
        } else {
            printf("Error reported by blockIdx=(%d %d %d)", report->_blockIdx.x, report->_blockIdx.y,
                   report->_blockIdx.z);
            printf("threadIdx=(%d %d %d)\n", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
        }
        if (file != nullptr)
            printf("file %s, line %d\n", file, line);
        exit(1);
    }
}

template<class params>
__global__
void parallel_factorize_kernel(cgbn_error_report_t *report,
                               cgbn_mem_t<params::BITS> n,
                               const unsigned *primes,
                               unsigned random_mul,
                               unsigned b_max,
                               unsigned b_start,
                               unsigned b_jump,
                               volatile bool *completed,
                               factor_result_t<params> *result) {
    typedef cgbn_context_t<params::TPI> context_t;
    typedef cgbn_env_t<context_t, params::BITS> env_t;
    typedef typename env_t::cgbn_t bn_t;

    if (*completed) return;

    const unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned instance = tid / params::TPI;
    const unsigned B = b_start + (b_jump * (blockIdx.x + 1)) * instance; // faster (other than linear) B growth ?
    if (B > b_max) return;

    const double log_b = log2((double) B);
    const unsigned log_b_ceil = ((unsigned) log_b) + 1;
    const unsigned prime_per_iter = params::BITS / log_b_ceil / 2;

    context_t bn_context(cgbn_report_monitor, report, instance);   // construct a context
    env_t bn_env(bn_context);                                  // construct an environment for big-int math

    unsigned primes_it;
    unsigned power;
    unsigned prime_i;

    bn_t N, a, d, e_sub, e, g, tmp;
    cgbn_load(bn_env, N, &n);
    cgbn_set_ui32(bn_env, a, (ULong) 2 + tid);
    cgbn_set_ui32(bn_env, d, 0);
    cgbn_set_ui32(bn_env, e, 0);
    cgbn_set_ui32(bn_env, e_sub, 0);
    cgbn_set_ui32(bn_env, g, 0);
    cgbn_set_ui32(bn_env, tmp, 0);

    // check NWD(a, N), if 0 then we have a factor else we can proceed with the algorithm
    cgbn_gcd(bn_env, d, a, N);
    if (cgbn_compare_ui32(bn_env, d, 1)) {
        *completed = true;
        cgbn_store(bn_env, &result->factor, d);
        result->b = B;
        return;
    }

    cgbn_set(bn_env, e, a); // e = a
    prime_i = (ULong) primes[0];
    for (primes_it = 0; prime_i <= B; primes_it += prime_per_iter) {
        if (*completed) return;
        cgbn_set_ui32(bn_env, e_sub, 1);

        for (unsigned i = 0; i < prime_per_iter && prime_i <= B; i++) {
            power = (unsigned) (log_b / log2((double) prime_i)); // p_pow_i = log(B) / log(p_i)
            cgbn_mul_ui32(bn_env, tmp, e_sub, (unsigned) pow((double) prime_i, power)); // e_sub *= p_i^p_pow_i
            cgbn_set(bn_env, e_sub, tmp);
            prime_i = primes[primes_it + i + 1];
        }
        cgbn_modular_power(bn_env, g, e, e_sub, N); // e = (e ** e_sub) % N - partial
        cgbn_set(bn_env, e, g);
    }

    if (!cgbn_equals_ui32(bn_env, e, 1)) {
        if (*completed) return;

        cgbn_sub_ui32(bn_env, g, e, 1); // g = e - 1
        cgbn_gcd(bn_env, d, g, N); // d = gcd(g, N)

        if (cgbn_compare_ui32(bn_env, d, 1) <= 0) {
            return;
        } else if (cgbn_compare(bn_env, d, N) >= 0) {
            return;
        } else { // factor found!
            *completed = true;
            cgbn_store(bn_env, &result->factor, d);
            result->b = B;
            return;
        }
    }
}

int cudaInitialize() {
    hipError_t err;
    int num;
    if (hipSuccess != (err = hipGetDeviceCount(&num))) {
        fprintf(stderr, "Cannot get number of CUDA devices\nError [%d]%s\n", (int) err, hipGetErrorString(err));
        return -1;
    };
    if (num < 1) {
        fprintf(stderr, "No CUDA devices found\n");
        return -1;
    };

    hipDeviceProp_t prop;
    int MaxDevice = -1;
    int MaxGflops = -1;
    for (int dev = 0; dev < num; dev++) {
        if (hipSuccess != (err = hipGetDeviceProperties(&prop, dev))) {
            fprintf(stderr, "Error getting device %d properties\nError [%d]%s\n", dev, (int) err,
                    hipGetErrorString(err));
            return -1;
        };
        int Gflops = prop.multiProcessorCount * prop.clockRate;
        printf("CUDA Device %d: %s Gflops %f Processors %d Threads/Block %d\n", dev, prop.name, 1e-6 * Gflops,
               prop.multiProcessorCount, prop.maxThreadsPerBlock);
        if (Gflops > MaxGflops) {
            MaxGflops = Gflops;
            MaxDevice = dev;
        };
    };
    printf("Fastest CUDA Device %d: %s\n", MaxDevice, prop.name);

    //  Print and set device
    if (hipSuccess != (err = hipGetDeviceProperties(&prop, MaxDevice))) {
        fprintf(stderr, "Error getting device %d properties\nError [%d]%s\n", MaxDevice, (int) err,
                hipGetErrorString(err));
        return -1;
    };
    hipSetDevice(MaxDevice);

    printf("TotalGlobalMem=%lu [MB]\n", (unsigned long) (prop.totalGlobalMem / 1024u / 1024u));
    printf("TotalConstMem=%lu [kB]\n", (unsigned long) (prop.totalConstMem / 1024u));
    printf("ClockRate=%d [MHz]\n", prop.clockRate / 1000);
    printf("MemoryClockRate=%d [MHz]\n", prop.memoryClockRate / 1000);

    printf("MaxTexture1D=%d\n", prop.maxTexture1D);
    printf("MaxTexture1DLinear=%u [KB]\n", prop.maxTexture1DLinear / 1024u);

    printf("MaxTexture2D=%d x %d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
    printf("MaxTexture2DLinear=%d x %d\n", prop.maxTexture2DLinear[0], prop.maxTexture2DLinear[1]);

    printf("\n");
    return 0;
}

void to_mpz(mpz_t r, uint32_t *x, uint32_t count) {
    mpz_import(r, count, -1, sizeof(uint32_t), 0, 0, x);
}

void from_mpz(mpz_t s, uint32_t *x, uint32_t count) {
    size_t words;

    if (mpz_sizeinbase(s, 2) > count * 32) {
        fprintf(stderr, "from_mpz failed -- result does not fit\n");
        exit(1);
    }

    mpz_export(x, &words, -1, sizeof(uint32_t), 0, 0, s);
    while (words < count)
        x[words++] = 0;
}

unsigned *allocate_primes(const unsigned prime_table[], const unsigned primes_num) {
    hipError_t err;

    unsigned *dev_primes;

    if (hipSuccess != (err = hipMalloc((void **) &dev_primes, primes_num * sizeof(prime_table[0])))) {
        fprintf(stderr, "Unable to allocate device prime table!\nError [%d]%s\n", (int) err, hipGetErrorString(err));
        return nullptr;
    }

    if (hipSuccess !=
        (err = hipMemcpy(dev_primes, prime_table, primes_num * sizeof(prime_table[0]), hipMemcpyHostToDevice))) {
        fprintf(stderr, "Unable to allocate device prime table!\nError [%d]%s\n", (int) err, hipGetErrorString(err));
        return nullptr;
    }

    return dev_primes;
}

int free_primes(unsigned *dev_primes) {
    if (dev_primes != nullptr) hipFree(dev_primes);
    return 0;
}

template<class params>
int parallel_factorize_param(mpz_t n,
                             const unsigned *gpu_primes_table,
                             const unsigned primes_num,
                             unsigned b_max,
                             unsigned b_start,
                             unsigned b_jump,
                             mpz_t *factor,
                             unsigned *b_found) {
    hipError_t err;
    size_t result_size = sizeof(factor_result_t<params>);

    bool completed = false;
    unsigned start = 0;
    cgbn_mem_t<params::BITS> gpu_n;
    factor_result_t<params> *gpu_result = nullptr;
    factor_result_t<params> cpu_result;
    bool *gpu_completed = nullptr;
    unsigned *gpu_start = nullptr;
    cgbn_error_report_t *report;

    if (
            (hipSuccess != (err = hipMalloc((void **) &gpu_result, result_size))) ||
            (hipSuccess != (err = hipMalloc((void **) &gpu_completed, sizeof(bool)))) ||
            (hipSuccess != (err = hipMalloc((void **) &gpu_start, sizeof(unsigned)))) ||
            (hipSuccess != (err = hipMemset(gpu_result, 0L, result_size))) ||
            (hipSuccess != (err = hipMemset(gpu_completed, false, sizeof(bool)))) ||
            (hipSuccess != (err = hipMemset(gpu_start, 0L, sizeof(unsigned)))) ||
            (hipSuccess != (err = cgbn_error_report_alloc(&report)))
            ) {
        fprintf(stderr, "Cannot allocate GPU memory!\nError [%d]%s\n", (int) err, hipGetErrorString(err));
        return -1;
    }

    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    from_mpz(n, gpu_n._limbs, params::BITS / 32);
    unsigned randomMul = 4123457; //+ 1000 * rand() + rand();

    unsigned blocks_num = (b_max * params::TPI) / (b_jump * THREADS_PER_BLOCK);
    unsigned threads_per_block = THREADS_PER_BLOCK;
    parallel_factorize_kernel<params><<<blocks_num, threads_per_block>>>(report, gpu_n, gpu_primes_table, randomMul, b_max,
                                                                         b_start, b_jump,
                                                                         gpu_completed,
                                                                         gpu_result);

    if (hipSuccess != (err = hipDeviceSynchronize()))
        fprintf(stderr, "Unable to synchronize device!\nError [%d]%s\n", (int) err, hipGetErrorString(err));

    CGBN_CHECK(report);

    if (hipSuccess != (err = hipMemcpy(&completed, gpu_completed, sizeof(bool), hipMemcpyDeviceToHost))) {
        fprintf(stderr, "Unable to retrieve finished flag from host!\nError [%d]%s\n", (int) err,
                hipGetErrorString(err));
        return -1;
    }

    if (hipSuccess != (err = hipMemcpy(&start, gpu_start, sizeof(start), hipMemcpyDeviceToHost))) {
        fprintf(stderr, "Unable to retrieve work position from host!\nError [%d]%s\n", (int) err,
                hipGetErrorString(err));
        return -1;
    }

    if (hipSuccess != (err = hipMemcpy(&cpu_result, gpu_result, result_size, hipMemcpyDeviceToHost))) {
        fprintf(stderr, "Unable to retrieve result from host!\nError [%d]%s\n", (int) err, hipGetErrorString(err));
        return -1;
    }

    to_mpz(*factor, cpu_result.factor._limbs, params::BITS / 32);
    printf("Found with B: %d\n", cpu_result.b);
    *b_found = cpu_result.b;

    if (gpu_result != nullptr) hipFree(gpu_result);
    if (gpu_completed != nullptr) hipFree(gpu_completed);
    if (gpu_completed != nullptr) cgbn_error_report_free(report);

    return 0;
}

int gpu_factorize(mpz_t n,
                  const unsigned int *primes_table,
                  const unsigned primes_num,
                  unsigned b_max,
                  unsigned b_start,
                  unsigned b_jump,
                  mpz_t *factor,
                  unsigned *b_found) {
    if (n->_mp_size < 2) {
        typedef pollard_params_t<4, 128> params;
        return parallel_factorize_param<params>(n, primes_table, primes_num, b_max, b_start, b_jump, factor, b_found);
    } else if (n->_mp_size < 4) {
        typedef pollard_params_t<8, 256> params;
        return parallel_factorize_param<params>(n, primes_table, primes_num, b_max, b_start, b_jump, factor, b_found);
    } else if (n->_mp_size < 8) {
        typedef pollard_params_t<16, 512> params;
        return parallel_factorize_param<params>(n, primes_table, primes_num, b_max, b_start, b_jump, factor, b_found);
    } else if (n->_mp_size < 16) {
        typedef pollard_params_t<32, 1024> params;
        return parallel_factorize_param<params>(n, primes_table, primes_num, b_max, b_start, b_jump, factor, b_found);
    } else {
        typedef pollard_params_t<32, 2048> params;
        return parallel_factorize_param<params>(n, primes_table, primes_num, b_max, b_start, b_jump, factor, b_found);
    }
}
